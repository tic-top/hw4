
#include <hip/hip_runtime.h>
#include <stdio.h>
#include <iostream>
#include <math.h>
#include <stdlib.h>
#include <string.h>

using namespace std;
static const long long n = 1000;
int t;

__global__ void MatrixUpdate(double *A, double *B)
{
    long long idx = blockIdx.x * blockDim.x + threadIdx.x;
    if (idx < n * n)
    {
        long long i = idx / n;
        long long j = idx % n;
        if (i == 0 || i == n - 1 || j == 0 || j == n - 1)
            B[idx] = A[idx];
        else
        {
            auto t1 = max(min(A[idx - 1 + n], A[idx + 1 + n]), min(A[idx - 1 - n], A[idx + 1 - n]));
            auto t2 = min(max(A[idx - 1 + n], A[idx + 1 + n]), max(A[idx - 1 - n], A[idx + 1 - n]));
            B[idx] = min(t1, t2) + A[idx];
        }
    }
}

__global__ void MatrixVerify1(double *A, double *C)
{
    long long idx = blockIdx.x * blockDim.x + threadIdx.x;
    if (idx == 0)
    {
        C[2] = A[37 * n + 48];
    }
}

__global__ void MatrixSum(double *A, double *C, long long *D)
{
    long long idx = blockIdx.x * blockDim.x + threadIdx.x;
    if (idx < n * n)
    {
        if (idx % (2 * D[0]) == 0 && idx + D[0] < n * n)
        {
            A[idx] += A[idx + D[0]];
        }
    }
}

__global__ void MatrixVerify2(double *A, double *C)
{
    long long idx = blockIdx.x * blockDim.x + threadIdx.x;
    if (idx == 0)
    {
        C[0] = A[0];
    }
}

int main(int argc, char *argv[])
{
    t = atoi(argv[1]);
    double *d_A;
    double *d_B;
    double *d_final;
    double *d_C;
    long long *d_D;
    auto size = n * n * sizeof(double);
    hipMalloc(&d_A, size);
    hipMalloc(&d_B, size);
    hipMalloc(&d_C, 3 * sizeof(double));
    hipMalloc(&d_D, sizeof(long long));
    double h_A[n * n];
    double h_C[3];

    for (double i = 0; i < n; i++)
    {
        for (double j = 0; j < n; j++)
        {
            h_A[(long long)i * n + (long long)j] = (1 + cos(2 * i) + sin(j)) * (1 + cos(2 * i) + sin(j));
        }
    }

    hipEvent_t start, stop;
    float ttime;
    hipEventCreate(&start);
    hipEventCreate(&stop);
    hipEventRecord(start, 0);
    hipEventSynchronize(start);
    hipMemcpy(d_A, h_A, size, hipMemcpyHostToDevice);
    for (int i = 0; i < t / 2; ++i)
    {
        MatrixUpdate<<<(n * n + 255) / 256, 256>>>(d_A, d_B);
        hipDeviceSynchronize();
        MatrixUpdate<<<(n * n + 255) / 256, 256>>>(d_B, d_A);
        hipDeviceSynchronize();
    }

    if (t % 2 == 1)
    {
        MatrixUpdate<<<(n * n + 255) / 256, 256>>>(d_A, d_B);
        hipDeviceSynchronize();
        d_final = d_B;
    }
    else
        d_final = d_A;

    MatrixVerify1<<<(n * n + 255) / 256, 256>>>(d_final, d_C);
    hipDeviceSynchronize();
    long long st = 1;
    while (st <= n * n)
    {
        hipMemcpy(d_D, &st, sizeof(long long), hipMemcpyHostToDevice);
        MatrixSum<<<(n * n + 255) / 256, 256>>>(d_final, d_C, d_D);
        hipDeviceSynchronize();
        st *= 2;
    }
    MatrixVerify2<<<(n * n + 255) / 256, 256>>>(d_final, d_C);
    hipDeviceSynchronize();
    hipMemcpy(h_C, d_C, 3 * sizeof(double), hipMemcpyDeviceToHost);
    hipEventRecord(stop, 0);
    hipEventSynchronize(stop);
    hipEventElapsedTime(&ttime, start, stop);
    cout << "sum: " << h_C[0] << "\n";
    cout << "A[37, 47] = " << h_C[2] << "\n";
    cout << "Time: " << ttime << " ms";
    hipFree(d_A);
    hipFree(d_B);
    hipFree(d_C);
    return 0;
}